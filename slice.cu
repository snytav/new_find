#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"
//#include <time.h>

#include "slice.h"

 Slice::Slice(unsigned int k)
   {
	length=k;
	NN=(k-1)/SIZE_OF_LONG_INT +1;
	blocks=min(NN,MAX_BLOCK);
    IT=(NN-1)/blocks+1;

    hipMalloc(&d_v,NN*sizeof(unsigned long long int));
    }

 void Slice::ASSIGN(Slice *X)
 {
	 assign_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
 }

 void Slice::AND(Slice *X)
  {
 	 and_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
  }

 void Slice::OR(Slice *X)
   {
  	 or_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
   }

 void Slice::XOR(Slice *X)
   {
  	 xor_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
   }

 void Slice::NOT()
   {
  	 not_kernel<<<blocks,1>>>(d_v,NN,IT);
   }

 void Slice::SET()
   {
  	 set_kernel<<<blocks,1>>>(d_v,NN,IT);
   }

 void Slice::CLR()
   {
  	 clr_kernel<<<blocks,1>>>(d_v,NN,IT);
   }

 unsigned int Slice::FND()
 {
 	// вычислить конфигурацию
 	unsigned int threads,it;
 	int *d_res,h_res;
 	hipMalloc(&d_res, sizeof(int));

 	{
 	   threads = min(MAX_THREADS,NN);
 	   it=(NN-1)/threads+1;
 //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
 	}

 	find_kernel<<<1,threads>>>(d_v,length,NN,it,d_res);
 	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
 	return h_res;
 }

 unsigned int Slice::NUMB()
 {
 	// вычислить конфигурацию
 	unsigned int threads,it;
 	int *d_res,h_res;
 	hipMalloc(&d_res, sizeof(int));

 	{
 	   threads = min(MAX_THREADS,NN);
 	   it=(NN-1)/threads+1;
 //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
 	}

 	tail_kernel<<<1,1>>>(d_v,length,NN);
 	numb_kernel<<<1,threads>>>(d_v,length,NN,it,d_res);
 	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
 	return h_res;
 }

 bool Slice::SOME()
 {
 	// вычислить конфигурацию
 	unsigned int threads,it;
 	int *d_res,h_res;
 	hipMalloc(&d_res, sizeof(int));

 	{
 	   threads = min(MAX_THREADS,NN);
 	   it=(NN-1)/threads+1;
 //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
 	}

 	tail_kernel<<<1,1>>>(d_v,length,NN);
 	some_kernel<<<1,threads>>>(d_v,NN,it,d_res);
 	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
 	return h_res==1;
 }

 bool Slice::ZERO()
  {
  	// вычислить конфигурацию
  	unsigned int threads,it;
  	int *d_res,h_res;
  	hipMalloc(&d_res, sizeof(int));

  	{
  	   threads = min(MAX_THREADS,NN);
  	   it=(NN-1)/threads+1;
  //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
  	}

  	tail_kernel<<<1,1>>>(d_v,length,NN);
  	zero_kernel<<<1,threads>>>(d_v,NN,it,d_res);
  	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
  	return h_res==1;
  }
