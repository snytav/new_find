#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"
//#include <time.h>

#include "slice.h"

 Slice::Slice(unsigned int k)
   {
	length=k;
	NN=(k-1)/SIZE_OF_LONG_INT +1;
	blocks=min(NN,MAX_BLOCK);
    IT=(NN-1)/blocks+1;

    hipMalloc(&d_v,NN*sizeof(unsigned long long int));
    }

 void Slice::ASSIGN(Slice *X)
 {
	 assign_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
 }

 void Slice::AND(Slice *X)
  {
 	 and_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
  }

 void Slice::OR(Slice *X)
   {
  	 or_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
   }

 void Slice::XOR(Slice *X)
   {
  	 xor_kernel<<<blocks,1>>>(d_v, X->get_device_pointer(),NN,IT);
   }

 void Slice::NOT()
   {
  	 not_kernel<<<blocks,1>>>(d_v,NN,IT);
   }

 void Slice::SET()
   {
  	 set_kernel<<<blocks,1>>>(d_v,NN,IT);
   }

 void Slice::CLR()
   {
  	 clr_kernel<<<blocks,1>>>(d_v,NN,IT);
   }

 unsigned int Slice::FND()
 {
 	// вычислить конфигурацию
 	unsigned int threads,it;
 	int *d_res,h_res;
 	hipMalloc(&d_res, sizeof(int));

 	{
 	   threads = min(MAX_THREADS,NN);
 	   it=(NN-1)/threads+1;
 //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
 	}

 	find_kernel<<<1,threads>>>(d_v,length,NN,it,d_res);
 	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
 	return h_res;
 }

 unsigned int Slice::NUMB()
 {
 	// вычислить конфигурацию
 	unsigned int threads,it;
 	int *d_res,h_res;
 	hipMalloc(&d_res, sizeof(int));

 	{
 	   threads = min(MAX_THREADS,NN);
 	   it=(NN-1)/threads+1;
 //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
 	}

 	tail_kernel<<<1,1>>>(d_v,length,NN);
 	numb_kernel<<<1,threads>>>(d_v,length,NN,it,d_res);
 	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
 	return h_res;
 }

 bool Slice::SOME()
 {
 	// вычислить конфигурацию
 	unsigned int threads,it;
 	int *d_res,h_res;
 	hipMalloc(&d_res, sizeof(int));

 	{
 	   threads = min(MAX_THREADS,NN);
 	   it=(NN-1)/threads+1;
 //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
 	}

 	tail_kernel<<<1,1>>>(d_v,length,NN);
 	some_kernel<<<1,threads>>>(d_v,NN,it,d_res);
 	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
 	return h_res==1;
 }

 bool Slice::ZERO()
  {
  	// вычислить конфигурацию
  	unsigned int threads,it;
  	int *d_res,h_res;
  	hipMalloc(&d_res, sizeof(int));

  	{
  	   threads = min(MAX_THREADS,NN);
  	   it=(NN-1)/threads+1;
  //	   printf("N=%d threads=%d,IT=%d \n",NN,threads,it);
  	}

  	tail_kernel<<<1,1>>>(d_v,length,NN);
  	zero_kernel<<<1,threads>>>(d_v,NN,it,d_res);
  	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
  	return h_res==1;
  }

 void __global__ digit_kernel(unsigned long long *w, unsigned long long *dig)
 {
 	dig[0]=__brevll(w[0]);
 //	dig[0]=w[0];
 }

 unsigned long long int Slice::ToDigit()
 { unsigned long long *d_dig1,res=0;
 	if (NN==1)
 	{  hipMalloc(&d_dig1,sizeof(unsigned long long));
 		digit_kernel<<<1,1>>>(d_v,d_dig1);
 		hipMemcpy(&res,d_dig1,sizeof(unsigned long long),hipMemcpyDeviceToHost);
 		res>>=(64-length);
 	}
 	return res;
 }
 void Slice::FromDigit(unsigned long long dig)
 {	 unsigned long long *d_dig1;
 		if (NN==1)
 		{   dig<<=(64-length);
 			hipMalloc(&d_dig1,sizeof(unsigned long long));
 			hipMemcpy(d_dig1,&dig,sizeof(unsigned long long),hipMemcpyHostToDevice);
 			digit_kernel<<<1,1>>>(d_dig1,d_v);
 		}
 }


 void __global__ print_kernel(LongPointer d_v, char* d_str, unsigned int length,unsigned int N1,unsigned int it)
 {
	 unsigned long long int tmp,one=1;
	 unsigned int index=(blockIdx.x*blockDim.x+threadIdx.x)*it;
	 for(int i=0; i<it;i++)
		if (index+i<N1)
		{
			tmp=d_v[index+i];
			for (int j=0;j< SIZE_OF_LONG_INT;j++)
			{
				d_str[(index+i)*SIZE_OF_LONG_INT+j] =(tmp&one)?'1':'0';
				tmp=tmp>>1;
				if(((index+i)*SIZE_OF_LONG_INT+j)==length) d_str[length]=0;
			}
		}
 }

 void Slice::print(char *label)
 { char *d_str, *str;
 	 hipMalloc(&d_str,NN*SIZE_OF_LONG_INT*sizeof(char));
 	 str=new char[NN*SIZE_OF_LONG_INT];
 	 print_kernel<<<blocks,1>>>(d_v,d_str,length,NN,IT);
 	hipMemcpy(str,d_str,NN*SIZE_OF_LONG_INT*sizeof(char),hipMemcpyDeviceToHost);
 	printf("%s \n%s\n",label,str);

 }

 void Slice::fprint(char *label)
  { char *d_str, *str;
  	 hipMalloc(&d_str,NN*SIZE_OF_LONG_INT*sizeof(char));
  	 str=new char[NN*SIZE_OF_LONG_INT];
  	 print_kernel<<<blocks,1>>>(d_v,d_str,length,NN,IT);
  	hipMemcpy(str,d_str,NN*SIZE_OF_LONG_INT*sizeof(char),hipMemcpyDeviceToHost);

  	FILE * pFile;
  	char fname[30]{0};
  	strcat(fname,label);
  	strcat(fname,".dat");
  	pFile = fopen (fname,"w");
  	fprintf(pFile,"%s (%d)\n%s\n",label,length,str);
  	fclose (pFile);
  }
