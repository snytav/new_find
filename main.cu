#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <time.h>
#include "slice.h"
#include "cuPrintf.cuh"
#include "cuPrintf.cu"

#define MAX 999999
#define N 100000
#define L  N*64-32

unsigned int FND(unsigned long long *d_v)
{
	// вычислить конфигурацию для find_simple
	unsigned int N1 =N,threads,it;
	int *d_res,h_res;
	hipMalloc(&d_res, sizeof(int));

//	for (N1=512;N1<1000000;N1=N1<<1)
	{
	   threads = min(MAX_THREADS,N1);
	   it=(N1-1)/threads+1;
	   printf("N=%d threads=%d,IT=%d \n",N1,threads,it);
	}
	numb_kernel<<<1,threads>>>(d_v,N*64-32,N1,it,d_res);
	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
	printf("NUMB=%d\n",h_res);

	find_kernel<<<1,threads>>>(d_v,(N*64),N1,it,d_res);
	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
	return h_res;
}


int main()
{
 /*   unsigned long long h_v[N];/* = {0xABCDABCDABCD0000, 0x0F08000800080070,
                                0xABCDABCDAB900000, 0x0F08000800080700,
                                0xABCDABCDABC80000, 0x0F08000807000000,
                                0xABCDABCDAB001000, 0x0F08000800080500 };*/
/*
    unsigned long long* d_v;
//  int *d_res,h_res;
//   hipMalloc(&d_res, sizeof(int));

    for (int i = 0; i < N; i++)
    {
        h_v[i] = (i >0) ? 0x8000000000000008 : 0;//rand() % MAX + 1;
        int sh = rand() % 32 + 1;
        //h_v[i] <<= sh;
        //printf("%d %30lx shift %d \n",i,h_v[i],sh);
    }
    hipMalloc(&d_v, N * sizeof(unsigned long long));

    hipMemcpy(d_v, h_v, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
 // печать с устройства
    cudaPrintfInit();

    printf("FND= %d \n",FND(d_v));

    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
*/
    Slice X(L);
    X.fprint("X1");
    X.SET();
    X.fprint("X2");
    printf("%d %d",X.length,X.NUMB());
    return 0;
}
 
