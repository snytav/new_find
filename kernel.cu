#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <time.h>
#define MAX 999999
#include "cuPrintf.cuh"
#include "cuPrintf.cu"
#define SIZE_OF_LONG_INT 64
//максимальная длина массива из длинных целых (для буферного массива я ядре find)
#define N 100000

/* В дальнейшем метод класса Slice
 * внутри функции доступны
 * unsigned int length - длина слайса в битах
 * unsigned int N - длина слайса в 64-х разрядных целых
 * unsigned int IT - количество элементов, обрабатываемых одним потоком (для N>1024)
 *
 * константы
 *  #define MAX_THREADS 1024
 */
#define MAX_THREADS 1024
void __global__ find_kernel(unsigned long long* d_v, unsigned int length,unsigned int N1,unsigned int it, int* res)
{
    __shared__ unsigned int res_by_thread[MAX_THREADS];
    unsigned int local_1st_nonzero,local_it_1st_nonzero,tmp;
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int active_threads=gridDim.x * blockDim.x;

    // res_by_thread[n] для it элементов:
    local_1st_nonzero=SIZE_OF_LONG_INT*N+1;
    for(int i=0;i<it;i++){
    	local_it_1st_nonzero = __ffsll(d_v[n*it+i]);  // первая единица в слове
    //номер этой первой единицы глобальный (по всему массиву) начиная справа
    	tmp=(local_it_1st_nonzero!=0)?(local_it_1st_nonzero+(n*it+i)*SIZE_OF_LONG_INT): (SIZE_OF_LONG_INT*N+1);
    	local_1st_nonzero=min(local_1st_nonzero,tmp);
    }
    res_by_thread[n]=local_1st_nonzero;

   while(active_threads>0)
    {
        __syncthreads();
        active_threads=active_threads>>1;
        if (n < active_threads)
        {
            res_by_thread[n] = min(res_by_thread[n], res_by_thread[active_threads+n]);
        }
    }
    if (n==0)
    {
    	*res = res_by_thread[0];
        if (*res>length) *res = 0;
    }
}
unsigned int FND(unsigned long long *d_v)
{
	// вычислить конфигурацию для find_simple
	unsigned int N1 =N,threads,it;
	int *d_res,h_res;
	hipMalloc(&d_res, sizeof(int));

//	for (N1=512;N1<1000000;N1=N1<<1)
	{
	   threads = min(MAX_THREADS,N1);
	   it=(N1-1)/threads+1;
	   printf("N=%d threads=%d,IT=%d \n",N1,threads,it);
	}
	find_kernel<<<1,threads>>>(d_v,N*64,N1,it,d_res);
	hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);
	return h_res;
}

int main()
{
    unsigned long long h_v[N];/* = {0xABCDABCDABCD0000, 0x0F08000800080070,
                                0xABCDABCDAB900000, 0x0F08000800080700,
                                0xABCDABCDABC80000, 0x0F08000807000000,
                                0xABCDABCDAB001000, 0x0F08000800080500 };*/
    unsigned long long* d_v;
//  int *d_res,h_res;
//   hipMalloc(&d_res, sizeof(int));

    for (int i = 0; i < N; i++)
    {
        h_v[i] = (i == (800)) ? 0x8000000000000000 : 0;//rand() % MAX + 1;
        int sh = rand() % 32 + 1;
        //h_v[i] <<= sh;
        //printf("%d %30lx shift %d \n",i,h_v[i],sh);
    }
    hipMalloc(&d_v, N * sizeof(unsigned long long));

    hipMemcpy(d_v, h_v, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
 /* печать с устройства
    cudaPrintfInit();

    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
*/

    printf("FND= %d \n",FND(d_v));
    return 0;
}
