#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "cuPrintf.cuh"
#include "cuPrintf.cu"

__global__ void test()
{
      cuPrintf("test");
}


int main()
{

    cudaPrintfInit();
    test << <1, 10 >> > ();
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();



    return 0;
}
