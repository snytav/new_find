#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>

#include "cuPrintf.cuh"
#include "cuPrintf.cu"

#define SIZE_OF_LONG_INT 64

//максимальная длина массива из длинных целых (для буферного массива я ядре find)
#define N 8
__global__ void test()
{
      cuPrintf("test");
}

int  __device__ gap(int level)
{
    //cuPrintf("gap entered level %d\n",level);
    int d2 = (int)pow(2, level);
    int g = (gridDim.x * blockDim.x) / d2;
    //cuPrintf("gap %d gridDim.x %d blockDim.x %d denom %d level %d\n",
      //        g,   (int)gridDim.x,   (int)blockDim.x,d2,level);

    return g;
}

unsigned int __device__ get_num_thread_to_compare(int level)
{
   
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;

    return gap(level) + n;
}

int __device__ active_thread(int level)
{
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    return (n < gap(level));
}


//считается что size равен количеству потоков
void __global__ find(unsigned long long* d_v, int size, int* res)
{

    __shared__ int res_by_thread[N];
    int local_1st_nonzero;
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    local_1st_nonzero = __ffsll(d_v[n]);  // первая единица в слове
    int levels = int(log((double)(N)) / log(2.0));

   
   
    //номер этой первой единицы глобальный (по всему массиву) начиная справа
    res_by_thread[n] = local_1st_nonzero + SIZE_OF_LONG_INT * (size-n-1);
    cuPrintf("n %d res by thread local %d global %d\n",n, 
        local_1st_nonzero,res_by_thread[n]);
    

    cuPrintf("reduction levels %d active %d gap %d \n",levels, active_thread(levels),gap(levels));
    //return;
    for (int l = 1; l <= levels;l++)
    {
        cuPrintf("l in loop %d activ %d\n",l, active_thread(l));
        if (active_thread(l))
        {
            unsigned int m = get_num_thread_to_compare(l);
            cuPrintf("level %d gap %u compare with %u res_by_thread[n] %d res_by_thread[m] %d \n",
                l,gap(l), m, res_by_thread[n], res_by_thread[m]);
            res_by_thread[n] = min(res_by_thread[n],
                                        res_by_thread[m]);
            cuPrintf("res_by_thread[n] %d res_by_thread[m] %d \n",
                      res_by_thread[n],   res_by_thread[m]);

        }
    }

   
    *res = res_by_thread[n];
    cuPrintf("global min %d\n",*res);
}


int main()
{
    unsigned long long h_v[] = {0xABCDABCDABCD0000, 0x0F08000800080070,
                                0xABCDABCDAB900000, 0x0F08000800080700,
                                0xABCDABCDABC80000, 0x0F08000807000000,
                                0xABCDABCDAB001000, 0x0F08000800080500 };
    unsigned long long* d_v;
    int *d_res,h_res;

    hipMalloc(&d_v, N * sizeof(unsigned long long));
    hipMalloc(&d_res, sizeof(int));
    hipMemcpy(d_v, h_v, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
    cudaPrintfInit();
//    gap << <1, N >> > (2);
    find << <1, N >> > (d_v,N,d_res);
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
    hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);




    return 0;
}
