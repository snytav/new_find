#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>

#include "cuPrintf.cuh"
#include "cuPrintf.cu"

#define SIZE_OF_LONG_INT 64

//максимальная длина массива из длинных целых (для буферного массива я ядре find)
#define N 2
__global__ void test()
{
      cuPrintf("test");
}

unsigned int __device__ gap(int level)
{
    int denom = (int)pow(2, level);
    unsigned int g = (gridDim.x * blockDim.x) / level / denom;
    return g;
}

unsigned int __device__ get_num_thread_to_compare(int level)
{
   
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;

    return gap(level) + n;
}

int __device__ active_thread(int level)
{
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    return (n < gap(level));
}


//считается что size равен количеству потоков
void __global__ find(unsigned long long* d_v, int size, int* res)
{
    __shared__ int res_by_thread[N];
    int local_1st_nonzero;
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    local_1st_nonzero = __ffsll(d_v[n]);  // первая единица в слове
    int levels = int(log((double)N) / log(2.0));
   
    //номер этой первой единицы глобальный (по всему массиву) начиная справа
    res_by_thread[n] = local_1st_nonzero + SIZE_OF_LONG_INT * (size-n-1);
    cuPrintf("n %d res by thread local %d global %d\n",n, 
        local_1st_nonzero,res_by_thread[n]);
    

    cuPrintf("reduction levels %d active %d gap %d \n",levels, active_thread(1),gap(1));
   
    for (int l = 1; l <= levels; l++)
    {
        if (active_thread(l))
        {
            unsigned int m = get_num_thread_to_compare(l);
            cuPrintf("level %d gap %u compare with %u \n", l,gap(l), m);
            res_by_thread[n] = min(res_by_thread[n],
                                        res_by_thread[m]);
            cuPrintf("res_by_thread[n] %d res_by_thread[m] %d \n",
                      res_by_thread[n],   res_by_thread[m]);

        }
    }

   
//    *res = minp;
    cuPrintf("global min %d\n",*res);
}


int main()
{
    unsigned long long h_v[] = {0xABCDABCDABCD0000, 0x0F08000800080070};
    unsigned long long* d_v;
    int *d_res,h_res;

    hipMalloc(&d_v, N * sizeof(unsigned long long));
    hipMalloc(&d_res, sizeof(int));
    hipMemcpy(d_v, h_v, N * sizeof(unsigned long long), hipMemcpyHostToDevice);
    cudaPrintfInit();
    find << <1, N >> > (d_v,N,d_res);
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
    hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);




    return 0;
}
