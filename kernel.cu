#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"
//#include <time.h>
#include "kernel.h"


//максимальная длина массива из длинных целых (для буферного массива я ядре find)


/* В дальнейшем метод класса Slice
 * внутри функции доступны
 * unsigned int length - длина слайса в битах
 * unsigned int N - длина слайса в 64-х разрядных целых
 * unsigned int IT - количество элементов, обрабатываемых одним потоком (для N>1024)
 *
 * константы
 *  #define MAX_THREADS 1024
 */

void __global__ find_kernel(unsigned long long* d_v, unsigned int length,unsigned int N1,unsigned int it, int* res)
{
    __shared__ unsigned int res_by_thread[MAX_THREADS];
    unsigned int local_1st_nonzero,local_it_1st_nonzero,tmp;
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int active_threads=gridDim.x * blockDim.x;

    // res_by_thread[n] для it элементов:
    local_1st_nonzero=SIZE_OF_LONG_INT*N1+1;
    for(int i=0;i<it;i++){
    	local_it_1st_nonzero = __ffsll(d_v[n*it+i]);  // первая единица в слове
    //номер этой первой единицы глобальный (по всему массиву) начиная справа
    	tmp=(local_it_1st_nonzero!=0)?(local_it_1st_nonzero+(n*it+i)*SIZE_OF_LONG_INT): (SIZE_OF_LONG_INT*N1+1);
    	local_1st_nonzero=min(local_1st_nonzero,tmp);
    }
    res_by_thread[n]=local_1st_nonzero;

   while(active_threads>1)
    {
        __syncthreads();
        active_threads=active_threads>>1;
        if (n < active_threads)
        {
            res_by_thread[n] = min(res_by_thread[n], res_by_thread[active_threads+n]);
        }
    }
    if (n==0)
    {
    	*res = res_by_thread[0];
        if (*res>length) *res = 0;
    }
}

void __global__ some_kernel(unsigned long long* d_v,unsigned int N1,unsigned int it, int*res)
{
	__shared__ unsigned int tmp;
	unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
	if (n==0) tmp=0;
	__syncthreads();

	int i=0;
	while((!tmp)&&(i<it))
	{
		if(d_v[n*it+i]>0) tmp=1;
		i++;
	}
	__syncthreads();
	 if (n==0)
		 *res = tmp;
}

void __global__ zero_kernel(unsigned long long* d_v,unsigned int N1,unsigned int it, int*res)
{
	__shared__ unsigned int tmp;
	unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
	if (n==0) tmp=1;
	__syncthreads();

	int i=0;
	while((tmp)&&(i<it))
	{
		if(d_v[n*it+i]>0) tmp=0;
		i++;
	}
	__syncthreads();
	 if (n==0)
		 *res = tmp;
}

void __global__ numb_kernel(unsigned long long* d_v, unsigned int length,unsigned int N1,unsigned int it, int* res)
{
    __shared__ unsigned int res_by_thread[MAX_THREADS];
    unsigned int tmp;
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int active_threads=gridDim.x * blockDim.x;

    unsigned long long int tail;


    // res_by_thread[n] для it элементов:
    tmp=0;
    for(int i=n*it;i<(n+1)*it;i++){
    	if(i==(N1-1))
    	{   tail=1;
    		tail=(tail<<(length%SIZE_OF_LONG_INT))-1;
 //   		printf("length=%d,остаток= %d \t",length,length%SIZE_OF_LONG_INT);
 //   		printf("%lx %d \n",tail,tail==0);
            if (tail==0)
            {
            	tail=~0;
 //           	printf("tail2=%llu \t",tail);
            }
 //   		printf("%d:\t d_v[%d]=%lx tail=%lx ",N1,i,d_v[i],tail);
    		d_v[i]=d_v[i]&tail;
//    		printf(":%lx\n",d_v[i]);
    	}
    	tmp += __popcll(d_v[i]);  // первая единица в слове
    }
    res_by_thread[n]=tmp;

   while(active_threads>1)
    {
        __syncthreads();
        active_threads=active_threads>>1;
        if (n < active_threads)
        {
            res_by_thread[n] = res_by_thread[n]+ res_by_thread[active_threads+n];
        }
    }
    if (n==0)
    {
    	*res = res_by_thread[0];

    }
}


